#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "lodepng.h"
#include <hip/hip_runtime.h>

__global__
void invert_bit_kernel(unsigned char* img) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char a = ~img[i];
  img[i] = a;
}

int main( int argc, char ** argv){

  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";
  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);

  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  unsigned char *cudaImage;
  unsigned int size = height * width * 3;

  hipEvent_t startHostToDevice, stopHostToDevice;
  hipEvent_t startDeviceToHost, stopDeviceToHost;
  hipEvent_t start, stop;
  hipEventCreate(&startHostToDevice);
  hipEventCreate(&startDeviceToHost);
  hipEventCreate(&stopHostToDevice);
  hipEventCreate(&stopDeviceToHost);
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float timing, toDevice, fromDevice;

  hipEventRecord(start,0);

  /* Allocate and copy stuff to device */
  hipMalloc((void**)&cudaImage, size);

  hipEventRecord(startHostToDevice,0);
  hipMemcpy(cudaImage, image, size, hipMemcpyHostToDevice);
  hipEventRecord(stopHostToDevice,0);
  hipEventSynchronize(stopHostToDevice);
  hipEventElapsedTime(&toDevice, startHostToDevice, stopHostToDevice);

  /* Maximum number of threads for the its-015 GPUs */
  int nThreads = 1024;
  int nBlocks = size / nThreads;

  // Do work
  invert_bit_kernel<<<nBlocks, nThreads>>>(cudaImage);

  /* Get stuff from device to host */
  hipEventRecord(startDeviceToHost,0);
  hipMemcpy(image, cudaImage, size, hipMemcpyDeviceToHost);
  hipEventRecord(stopDeviceToHost,0);
  hipEventSynchronize(stopDeviceToHost);
  hipEventElapsedTime(&fromDevice, startDeviceToHost, stopDeviceToHost);

  hipFree(cudaImage);

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&timing, start, stop);

  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image, width, height);

  std::cout<<"\n\nElapsed Time To Device = "<<toDevice<<" ms";
  std::cout<<"\n\nElapsed Time From Device = "<<fromDevice<<" ms";
  std::cout<<"\n\nElapsed Time Total = "<<timing<<" ms";

  return 0;
}

