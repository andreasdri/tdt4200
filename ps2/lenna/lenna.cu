#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include "lodepng.h"


__global__
void invert_bit_kernel(unsigned char* img) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned char a = ~img[i];
  img[i] = a;
}

int main( int argc, char ** argv){

  size_t pngsize;
  unsigned char *png;
  const char * filename = "lenna512x512_inv.png";
  /* Read in the image */
  lodepng_load_file(&png, &pngsize, filename);

  unsigned char *image;
  unsigned int width, height;
  /* Decode it into a RGB 8-bit per channel vector */
  unsigned int error = lodepng_decode24(&image, &width, &height, png, pngsize);

  /* Check if read and decode of .png went well */
  if(error != 0){
      std::cout << "error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  unsigned char *cudaImage;
  unsigned int size = height * width * 3;

  /* Allocate and copy stuff to device */
  hipMalloc((void**)&cudaImage, size);
  hipMemcpy(cudaImage, image, size, hipMemcpyHostToDevice);

  /* Maximum number of threads for the its-015 GPUs */
  int nThreads = 1024;
  int nBlocks = size / nThreads;

  // Do work
  invert_bit_kernel<<<nBlocks, nThreads>>>(cudaImage);

  /* Get stuff from device to host */
  hipMemcpy(image, cudaImage, size, hipMemcpyDeviceToHost);
  hipFree(cudaImage);

  /* Save the result to a new .png file */
  lodepng_encode24_file("lenna512x512_orig.png", image, width, height);

  return 0;
}

